#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
using namespace std;

void check(hipError_t e)
{
	if (e != hipSuccess)
	{
		printf(hipGetErrorString(e));
	}
}



// Kernel function to add the elements of two arrays
__global__
void runningSum(int n, float *x)
{
	int id = threadIdx.x;
	
	for (int i = 1; i < n; i = i * 2)
	{
		if(id+i < n)
		x[id+i] += x[id] ;
		//__syncthreads();
	}
	
}

int main(void)
{
	int N = 8;
	float *x, *y;

	// Allocate Unified Memory � accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = i+1;
		cout<<x[i] <<"\t";
	}
	cout<<"\n";
	// Run kernel on 1M elements on the GPU
	runningSum<<<1, N>>>(N, x);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	for (int i = 0; i < N; i++)
		cout << x[i] <<"\t";

	// Free memory
	hipFree(x);
	hipFree(y);
	getchar();
	return 0;
}